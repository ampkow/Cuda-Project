// nvgraph_assign.cu -- NVGraph  Programming Assignment
// Adam Piorkowski
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nvgraph.h"

// __host__ allows to run on device and host and void code duplication
// taken from global_memory.cu
__host__ hipEvent_t get_time(void)
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
	return time;
}

 // Runs BFS on a generated graph and prints out
 // distances
float run_nvgraph_search(int num_elements)
{
    // Graph is in CSR format
    hipEvent_t start = get_time();

    int verticies = num_elements;
    int num_edges = num_elements;

    printf("Num_edges %d\n", num_edges);
    int vertex_numsets = 2; 

    // Create Verticies
    int offsets_size_in_bytes = sizeof(int) * (verticies + 1);
    int *host_offsets = (int *) malloc(offsets_size_in_bytes); 

    int dest_size_in_bytes = sizeof(int) * (num_edges);
    int *host_dest = (int *) malloc(dest_size_in_bytes); 

    for (int offsetIter = 0; offsetIter < verticies; offsetIter++)
    {
        host_offsets[offsetIter] = offsetIter;
    }

    // Set Up Destinations
    for (int destIter = 0; destIter < num_edges; destIter++)
    {
        host_dest[destIter] = host_offsets[destIter + 1];
    }
    
    // Last Value of Offsets equal number of edges in graph
    host_offsets[verticies] = num_edges;

    // holds results
    int *host_distances = (int *) malloc(offsets_size_in_bytes);

    // nvgraph values
    nvgraphHandle_t d_graph_handle;
    nvgraphGraphDescr_t d_desc;
    nvgraphCSRTopology32I_t d_input;
    hipDataType* dimT;
    int distances_index = 0;
    int predecessors_index = 1;
    dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    dimT[distances_index] = HIP_R_32I;
    dimT[predecessors_index] = HIP_R_32I;

    //Create Graph
    nvgraphStatus_t status = nvgraphCreate (&d_graph_handle);
    if ((int)status != 0)   
    {
        printf("ERROR nvgraphCreate: %d\n", status);
        exit(0);
    }

    status = nvgraphCreateGraphDescr (d_graph_handle, &d_desc);
    if ((int)status != 0)   
    {
        printf("ERROR nvgraphCreateGraphDescr: %d\n", status);
        exit(0);
    }

    // Set graph properties
    d_input = (nvgraphCSRTopology32I_t) malloc(sizeof(struct nvgraphCSRTopology32I_st));
    d_input->nvertices = verticies;
    d_input->nedges = num_edges;
    d_input->source_offsets = host_offsets;
    d_input->destination_indices = host_dest;

    status = nvgraphSetGraphStructure(d_graph_handle, d_desc, (void*)d_input, NVGRAPH_CSR_32);
    if ((int)status != 0)   
    {
        printf("ERROR nvgraphSetGraphStructure: %d\n", status);
        exit(0);
    }
    
    status = nvgraphAllocateVertexData(d_graph_handle, d_desc, vertex_numsets, dimT);

    if ((int)status != 0)   
    {
       printf("ERROR nvgraphAllocateVertexData: %d\n", status);
       exit(0);
    }

    hipEvent_t create_graph = get_time();

    int starter_vert = 1;
    // Setting the traverse param
    nvgraphTraversalParameter_t traversal_param;
    status = nvgraphTraversalParameterInit(&traversal_param);
    if ((int)status != 0)   
    {
       printf("ERROR nvgraphTraversalParameterInit: %d\n", status);
       exit(0);
    }

    status = nvgraphTraversalSetDistancesIndex(&traversal_param, distances_index); 
    if ((int)status != 0)   
    {
       printf("ERROR nvgraphTraversalSetDistancesIndex: %d\n", status);
       exit(0);
    }
   
    status = nvgraphTraversalSetPredecessorsIndex(&traversal_param, predecessors_index);
    if ((int)status != 0)   
    {
       printf("ERROR nvgraphTraversalSetPredecessorsIndex: %d\n", status);
       exit(0);
    }

    status = nvgraphTraversalSetUndirectedFlag(&traversal_param, false);
    if ((int)status != 0)   
    {
       printf("ERROR nvgraphTraversalSetUndirectedFlag: %d\n", status);
       exit(0);
    }

    printf("Graph Traverse set \n");
    
    status = nvgraphTraversal(d_graph_handle, d_desc, NVGRAPH_TRAVERSAL_BFS, &starter_vert, traversal_param);
    if ((int)status != 0)   
    {
       printf("ERROR nvgraphTraversal: %d\n", status);
       exit(0);
    }
    
    hipEvent_t traverse_time = get_time();

    // Get result
    status = nvgraphGetVertexData(d_graph_handle, d_desc, (void*)host_distances, distances_index);
    if ((int)status != 0)   
    {
       printf("ERROR nvgraphTraversal: %d\n",status);
       exit(0);
    }

    hipEvent_t data_retr_time = get_time();

    // Print distances for every verticies
    for (int iter1 = 0; iter1 < verticies; iter1++)  
    {
        printf("Distance to vertex %d: %i\n", iter1, host_distances[iter1]);
    }

    free(dimT);
    free(d_input);
    nvgraphDestroyGraphDescr (d_graph_handle, d_desc);
    nvgraphDestroy (d_graph_handle);

    
    hipEvent_t end = get_time();
    hipEventSynchronize(end);

    free(host_offsets);
    free(host_dest);
    free(host_distances);

    float graph_alloc = 0.0;
    hipEventElapsedTime(&graph_alloc, start, create_graph);

    float traverse_setup = 0.0;
    hipEventElapsedTime(&traverse_setup, create_graph, traverse_time);

    float result_ret_time = 0.0;
    hipEventElapsedTime(&result_ret_time, traverse_time, data_retr_time);

    float total_time;
    hipEventElapsedTime(&total_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(create_graph);
    hipEventDestroy(traverse_time);
    hipEventDestroy(data_retr_time);
    hipEventDestroy(end);

    printf("Time to create graph in memory  %f \n", graph_alloc);
    printf("Run time traverse setup and BFS %f \n", traverse_setup);
    printf("Time to copy memory back to host %f \n", result_ret_time);

    return total_time;
}

int main(int argc, char** argv)
{
	// read command line arguments
    int elements     = 256;

    if (argc >= 2) 
    {
		elements = atoi(argv[1]);
	}

    printf("Element test size is %d\n", elements);

    float run_time = run_nvgraph_search(elements);

    printf("Total run time for range test %f\n", run_time);

    printf("Succesful Run! --------------------------------\n");
    return 0;
}