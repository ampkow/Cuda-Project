#include "hip/hip_runtime.h"
// Contains GPU Cuda code that executes BFS algorithm
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>

// STL
#include <vector> 
#include <list> 
#include <iostream>

// Internal Headers
#include "KernelFunctions.h"

// taken from global_memory.cu, Creates event and records time
__host__ hipEvent_t get_time(void)
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
	return time;
}

/**
 * Runs BFS using Thrust Vectors and Thrust Library
 * graph       - contains all vertices and their edges
 * destination - destination vertex
 * source      - source vertex
 * totalEdges  - total number of edges in graph
 * 
**/
float RunBFSUsingThrust(std::vector<std::vector<int> > &graph,
                         int                            destination,
                         int                            source,
                         int                            totalEdges)
{
    std::list<int> nextVertList; 

    int vertexSize = graph.size();

    hipEvent_t start = get_time();

    // Device vectors, Allocate memory
    thrust::device_vector<int> d_visited(vertexSize);
    thrust::device_vector<int> d_predecessors(vertexSize);
    thrust::device_vector<int> d_distances(vertexSize);
    thrust::device_vector<int> d_path;

    // Initialize Values for vectors
    thrust::fill(d_visited.begin(), d_visited.end(), false);
    thrust::fill(d_distances.begin(), d_distances.end(), 0);
    thrust::fill(d_predecessors.begin(), d_predecessors.end(), -1);

    // Push source onto vertex list to iterate through
    nextVertList.push_back(source);
    d_visited[source] = true;

    hipEvent_t memFinished = get_time();

    bool foundDest = false;

    // BFS algorithm  
    while (!nextVertList.empty()) 
    {  
        int currVertIter = nextVertList.front();  // Current Vertex
        nextVertList.pop_front(); 

        // Amount of edges for current Vertex
        int edgeCount = graph.at(currVertIter).size();

        // need to populate with vector of edges for current vertex
        thrust::device_vector<int> d_edges(graph.at(currVertIter));

        // Iterate over edges of current vertex
        for (int edgeIter = 0; edgeIter < edgeCount; ++edgeIter)
        {
            // grab end of edge
            int nextVert = d_edges[edgeIter];

            // Check to see if edge has been visited yet
            if (d_visited[nextVert] == false) 
            {  
                d_visited[nextVert]      = true; 
                d_distances[nextVert]    = d_distances[currVertIter] + 1; 
                d_predecessors[nextVert] = currVertIter; 
    
                nextVertList.push_back(nextVert);
    
                // Return after reaching destination
                if (nextVert == destination) 
                {
                    foundDest = true;
                    break; 
                }
            }
        }
    }

    hipEvent_t bfsFinised = get_time();

    if (foundDest)
    {
        // Add destination to path
        d_path.push_back(destination);

        int pointer = destination;
        int pathSize = 1;

        // if predecessor was visited add to path 
        // for destination
        while (d_predecessors[pointer] != -1) 
        { 
            pathSize++;
            d_path.push_back(d_predecessors[pointer]); 
            pointer = d_predecessors[pointer];
        }       
              
        // printing path from source to destination 
        printf("\nShortest Path Length is %d\n", pathSize); 
        int count = 0;
        int iter = d_path.size() - 1;
        while (count < pathSize) 
        {
            std::cout << d_path[iter] << " "; 
            count++;
            iter--;
        }

        std::cout << std::endl;
    } 
    else
    {
        printf("Shortest Path not found to destination %d using Thrust library \n", destination); 
    }    

    hipEventSynchronize(bfsFinised);

    float memAllocTime;
    hipEventElapsedTime(&memAllocTime, start, memFinished);

    float kernelRunTime;
    hipEventElapsedTime(&kernelRunTime, memFinished, bfsFinised);
    
    float totalTime;
    hipEventElapsedTime(&totalTime, start, bfsFinised);

    // Clean Up Events
    hipEventDestroy(start);
    hipEventDestroy(memFinished);
    hipEventDestroy(bfsFinised);

    printf("Thrust Memory Allocation time %f\n", memAllocTime);
    printf("Thrust Kernenl Runtime %f \n", kernelRunTime);

    printf("\n");
    
    return totalTime; 
}


/**
 * Kernel Algorithm
 * Runs BFS on GPU by searching level by level
 * vertices     - list of vertices for GPU
 * edges        - list of edge destinations for GPU
 * distances    - stores distance from source to current vertex
 * predecessors - stores previous vertex
 * vertIndices  - list of start points for each vertices edges in edge list
 * edgeSize     - list of how many edges each vertex has
 * levels       - stores if current level of this vertex has been visited
 * visitedVert  - stores if vertices have been visited or not
 * foundDest    - to signal if destination has been found
 * numVert      - number of vertices in graph
 * destination  - destination vertex
 **/
__global__ void BFSLevels(int  *vertices,
                          int  *edges,
                          int  *distances,
                          int  *predecessors,
                          int  *vertIndices,
                          int  *edgeSize,
                          bool *levels,
                          bool *visitedVertices,
                          bool *foundDest,
                          int   numVert,
                          int   destination)
{
    // Grab ThreadID
    int thrID = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ bool destFound;
    destFound = false;

    if (thrID < numVert && !destFound)
    {
        int curVert = vertices[thrID];

        // Iterate through level if true
        if (levels[curVert])
        {
            levels[curVert]          = false;
            visitedVertices[curVert] = true;

            // Grab indexes for curVert edges in edge array
            int edgesEnd  = edgeSize[thrID] + vertIndices[thrID];  

            // Iterate through all edges for current vertex
            for (int edgeIter = vertIndices[thrID]; edgeIter < edgesEnd; ++edgeIter)
            {
                // Grab next Vertex at end of edge
                int nextVert = edges[edgeIter];

                // If it hasn't been visited store info 
                // for distance and predecessors and set level
                // to true for next level of vertices
                if (!visitedVertices[nextVert])
                {       
                    distances[nextVert] = distances[curVert] + 1;
                    levels[nextVert] = true;
                    predecessors[nextVert]  = curVert; 
      
                    // Set found destination to true and sync threads
                    if (nextVert == destination) 
                    {
                        *foundDest = true;
                        destFound  = true;
                        __syncthreads();
                    }
                }
            }
        }
    }    
}

/**
 * Runs BFS on GPU by searching level by level
 * vertices    - list of vertices for GPU
 * edges       - list of edge destinations for GPU
 * vertIndices - list of start points for each vertices edges in edge list
 * edgeSize    - list of how many edges each vertex has
 * destination - destination vertex
 * source      - source vertex
 **/
float BFSByLevel(std::vector<int> &vertices,
                 std::vector<int> &edges,
                 std::vector<int> &vertIndices,
                 std::vector<int> &edgeSize,
                 int               destination,
                 int               source)
{
    int numVertices = vertices.size();

    int arraySizeInBytes     = sizeof(int) * numVertices;
    int arraySizeInBytesBool = sizeof(bool) * numVertices;

    // Create Host Arrays to pass into kernel call for BFS
    bool *h_visitedVertices;
    bool *h_levels;
    int  *h_distances;
    int  *h_predecessors;

    h_distances        = (int *)  malloc(arraySizeInBytes); 
    h_predecessors     = (int *)  malloc(arraySizeInBytes);
    h_visitedVertices  = (bool *) malloc(arraySizeInBytesBool); 
    h_levels           = (bool *) malloc(arraySizeInBytesBool); 

    // Initializes arrays data
    for (int vertexIter = 0; vertexIter < numVertices; ++vertexIter)
    {
        h_visitedVertices[vertexIter] = false;
        h_levels[vertexIter]          = false;
        h_predecessors[vertexIter]    = -1;
    }

    h_levels[source]    = true;
    h_distances[source] = 0;

    hipEvent_t start = get_time();

    // Create device arrays
    bool *d_visitedVertices;
    bool *d_levels;
    int  *d_distances;
    int  *d_predecessors;

    // use thrust library for host vectors
    thrust::device_vector<int> d_vertices(vertices);
    thrust::device_vector<int> d_edges(edges);
    thrust::device_vector<int> d_vertIndices(vertIndices);
    thrust::device_vector<int> d_edgeSize(edgeSize);

    hipError_t result;

    // allocate other device arrays
    result  = hipMalloc((void**) &d_distances, arraySizeInBytes);
    if (result > 0)
    {
        printf("ERROR -- CUDAMALLOC failed to allocate memory\n");
        return 0.0;
    }

    result = hipMalloc((void**) &d_predecessors, arraySizeInBytes);
    if (result > 0)
    {
        printf("ERROR -- CUDAMALLOC failed to allocate memory\n");
        hipFree(d_distances);
        return 0.0;
    }

    result = hipMalloc((void**) &d_levels, arraySizeInBytesBool);
    if (result > 0)
    {
        printf("ERROR -- CUDAMALLOC failed to allocate memory\n");
        hipFree(d_distances);
        hipFree(d_predecessors);
        return 0.0;
    }

    result = hipMalloc((void**) &d_visitedVertices, arraySizeInBytesBool);
    if (result > 0)
    {
        printf("ERROR -- CUDAMALLOC failed to allocate memory\n");
        hipFree(d_distances);
        hipFree(d_predecessors);
        hipFree(d_levels);
        return 0.0;
    } 

    // Copy memory to device from host
    hipMemcpy(d_levels,           h_levels,           arraySizeInBytesBool, hipMemcpyHostToDevice);
    hipMemcpy(d_visitedVertices,  h_visitedVertices,  arraySizeInBytesBool, hipMemcpyHostToDevice);
    hipMemcpy(d_predecessors,     h_predecessors,     arraySizeInBytes,     hipMemcpyHostToDevice);
    hipMemcpy(d_distances,        h_distances,        arraySizeInBytes,     hipMemcpyHostToDevice);

    // Specify block count and threads, each vertex gets a thread
    int blockCount = 1;
    int numThreads = numVertices;

    // Use pinned memory to store whether destination was found or not
    bool *h_foundDest;

    result = hipHostAlloc((void**)&h_foundDest, sizeof(bool), hipHostMallocDefault);

    if (result > 0)
    {
        printf("ERROR -- CUDAHOSTALLOC failed to allocate memory\n");

        hipFree(d_distances);
        hipFree(d_predecessors);
        hipFree(d_levels);
        hipFree(d_visitedVertices);
        return 0.0;
    }

    h_foundDest = false;

    bool *d_foundDest;

    // Copy pinned memory to device
    result = hipMalloc((void**) &d_foundDest,    sizeof(bool));

    if (result > 0)
    {
        printf("ERROR -- CUDAMALLOC failed to allocate memory for destination\n");


        hipFree(d_distances);
        hipFree(d_predecessors);
        hipFree(d_levels);
        hipFree(d_visitedVertices);
        return 0.0;
    }

    hipMemcpy(d_foundDest,  &h_foundDest,  sizeof(bool), hipMemcpyHostToDevice);

    hipEvent_t memFinished = get_time();

    // Run BFS algorithm going through each level
    int runCount = 0;
    while (runCount < numVertices)
    {
        BFSLevels<<<blockCount, numThreads>>>(thrust::raw_pointer_cast(d_vertices.data()),
                                              thrust::raw_pointer_cast(d_edges.data()),
                                              d_distances,
                                              d_predecessors,
                                              thrust::raw_pointer_cast(d_vertIndices.data()),
                                              thrust::raw_pointer_cast(d_edgeSize.data()),
                                              d_levels,
                                              d_visitedVertices,
                                              d_foundDest,
                                              numVertices,
                                              destination);

        runCount++;
    }

    hipEvent_t kernelFinished = get_time();

    // Copy Back Results
    hipMemcpy(&h_foundDest, d_foundDest, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(h_distances, d_distances, arraySizeInBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_predecessors, d_predecessors, arraySizeInBytes, hipMemcpyDeviceToHost);

    // Free Device Memory
    hipFree(d_distances);
    hipFree(d_levels);
    hipFree(d_visitedVertices);
    hipFree(d_predecessors);

    hipFree(d_foundDest);

    hipEvent_t end = get_time();

    hipEventSynchronize(end);

    float totalTime;
    hipEventElapsedTime(&totalTime, start, end);

    float memAllocTime;
    hipEventElapsedTime(&memAllocTime, start, memFinished);

    float kernelRunTime;
    hipEventElapsedTime(&kernelRunTime, memFinished, kernelFinished);

    float copyMemBack;
    hipEventElapsedTime(&copyMemBack, kernelFinished, end);

    // Clean Up Events
    hipEventDestroy(start);
    hipEventDestroy(memFinished);
    hipEventDestroy(kernelFinished);
    hipEventDestroy(end);
    
    printf("GPU Memory Allocation time %f\n", memAllocTime);
    printf("GPU Kernenl Runtime %f \n", kernelRunTime);
    printf("GPU Copy Memory Back %f \n", copyMemBack);
  

    std::vector<int> path;

    if (h_foundDest)
    {
        // Push destination onto path
        path.push_back(destination);
        int pointer = destination;

        // Push each predecessor that was visited onto path
        while (h_predecessors[pointer] != -1) 
        { 
            path.push_back(h_predecessors[pointer]); 
            pointer = h_predecessors[pointer];
        }       
              
        // printing path from source to destination 
        printf("\nShortest Path Length is %zd\n", path.size()); 
        int count = 0;
        int iter = path.size() - 1;
        while (count < path.size()) 
        {
            std::cout << path[iter] << " "; 
            count++;
            iter--;
        }
        std::cout << std::endl;
    }
    else
    {
        printf("Shortest Path not found to destination %d using GPU BFS Search by levels  \n", destination); 
    }

    // Free Host Memory
    free(h_visitedVertices);
    free(h_levels);
    free(h_distances);
    free(h_predecessors);

    hipHostFree(h_foundDest);

    return totalTime;

}