#include "hip/hip_runtime.h"
// Make Kernel Functions inside of here
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>


#include "nvgraph.h"

// STL
#include <vector> 
#include <list> 
#include <iostream>

// Internal Headers
#include "KernelFunctions.h"


// __host__ allows to run on device and host and void code duplication
// taken from global_memory.cu
__host__ hipEvent_t get_time(void)
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
	return time;
}

__global__ void FindShortestPath(int *path,
                                 int *pred,
                                 int  dest)
{
    int pointer = dest;

    int pathSize = 0;

    while (pred[pointer] != -1) 
    { 
        pathSize++;
        path[pathSize] = pred[pointer]; 
        pointer = pred[pointer]; 
    } 

    // printing path from source to destination 
    printf("\n Shortest Path: \n"); 
    for (int iter = pathSize - 1; iter >= 0; iter--) 
    {
      printf("index %d ", path[iter]); 
    }

}
  

// Run BFS algorithm using thurst library
float RunBFSUsingThrust(std::vector<std::vector<int> > &graph,
                         int                             dest,
                         int                             source,
                         int                             totalEdges)
{
    std::list<int> queue; 

    int vertexSize = graph.size();

    thrust::device_vector<int> d_visited(vertexSize);
    thrust::device_vector<int> d_pred(vertexSize);
    thrust::device_vector<int> d_dist(vertexSize);
    thrust::device_vector<int> d_path(totalEdges);

    thrust::fill(d_visited.begin(), d_visited.end(), false);
    thrust::fill(d_dist.begin(), d_dist.end(), 0);
    thrust::fill(d_pred.begin(), d_pred.end(), -1);

    queue.push_back(source);
    d_visited[source] = true;

    bool foundDest = false;

    // BFS algorithm  
    while (!queue.empty()) 
    {  
        int queueIter = queue.front();  // Current Vertex
        queue.pop_front(); 

        int edgeCount = graph.at(queueIter).size();

        // need to populate with vector of edges for current vertex
        thrust::device_vector<int> d_edges(graph.at(queueIter));

        for (int iter0 = 0; iter0 < edgeCount; iter0++)
        {
            int nextVert = d_edges[iter0];
            if (d_visited[nextVert] == false) 
            { 
                printf("Element (%d, %d) = %d \n", queueIter, iter0, nextVert);
    
                d_visited[nextVert] = true; 
                d_dist[nextVert]    = d_dist[queueIter] + 1; 
                d_pred[nextVert]    = queueIter; 
    
                queue.push_back(nextVert);
    
                // Stop When finding destination
                if (nextVert == dest) 
                {
                    foundDest = true;
                    break; 
                }
            }
        }

        if (foundDest)
        {
            d_path.push_back(dest);

            int pointer = dest;

            int pathSize = 1;

            while (d_pred[pointer] != -1) 
            { 
               pathSize++;
               d_path.push_back(d_pred[pointer]); 
               pointer = d_pred[pointer];
            }       
              
            // printing path from source to destination 
            printf("\nShortest Path Length is %d\n", pathSize); 
            int count = 0;
            int iter = d_path.size() - 1;
            while (count < pathSize) 
            {
                std::cout << d_path[iter] << " "; 
                count++;
                iter--;
            }

            return true;
        }
    } 
    return false; 
}

 // Runs BFS on a generated graph and prints out
 float run_nvgraph_search(int numVerticies)
 {
     // Graph is in CSR format
     hipEvent_t start = get_time();
 
     int verticies = numVerticies;
     int num_edges = numVerticies;
 
     printf("Num_edges %d\n", num_edges);
     int vertex_numsets = 2; 
 
     // Create Verticies
     int offsets_size_in_bytes = sizeof(int) * (verticies + 1);
     int *host_offsets = (int *) malloc(offsets_size_in_bytes); 
 
     int dest_size_in_bytes = sizeof(int) * (num_edges);
     int *host_dest = (int *) malloc(dest_size_in_bytes); 
 
     for (int offsetIter = 0; offsetIter < verticies; offsetIter++)
     {
         host_offsets[offsetIter] = offsetIter;
     }
 
     // Set Up Destinations
     for (int destIter = 0; destIter < num_edges; destIter++)
     {
         host_dest[destIter] = host_offsets[destIter + 1];
     }

     // Last Value of Offsets equal number of edges in graph
     host_offsets[verticies] = num_edges;
 
     // holds results
     int *host_distances    = (int *) malloc(offsets_size_in_bytes);
     int *host_predecessors = (int *) malloc(offsets_size_in_bytes);
 
     // nvgraph values
     nvgraphHandle_t d_graph_handle;
     nvgraphGraphDescr_t d_desc;
     nvgraphCSRTopology32I_t d_input;
     hipDataType* dimT;
     int distances_index = 0;
     int predecessors_index = 1;
     dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
     dimT[distances_index] = HIP_R_32I;
     dimT[predecessors_index] = HIP_R_32I;
 
     //Create Graph
     nvgraphStatus_t status = nvgraphCreate (&d_graph_handle);
     if ((int)status != 0)   
     {
         printf("ERROR nvgraphCreate: %d\n", status);
         exit(0);
     }
 
     status = nvgraphCreateGraphDescr (d_graph_handle, &d_desc);
     if ((int)status != 0)   
     {
         printf("ERROR nvgraphCreateGraphDescr: %d\n", status);
         exit(0);
     }
 
     // Set graph properties
     d_input = (nvgraphCSRTopology32I_t) malloc(sizeof(struct nvgraphCSRTopology32I_st));
     d_input->nvertices = verticies;
     d_input->nedges = num_edges;
     d_input->source_offsets = host_offsets;
     d_input->destination_indices = host_dest;
 
     status = nvgraphSetGraphStructure(d_graph_handle, d_desc, (void*)d_input, NVGRAPH_CSR_32);
     if ((int)status != 0)   
     {
         printf("ERROR nvgraphSetGraphStructure: %d\n", status);
         exit(0);
     }
     
     status = nvgraphAllocateVertexData(d_graph_handle, d_desc, vertex_numsets, dimT);
 
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphAllocateVertexData: %d\n", status);
        exit(0);
     }
 
     hipEvent_t create_graph = get_time();
 
     int starter_vert = 1;
     // Setting the traverse param
     nvgraphTraversalParameter_t traversal_param;
     status = nvgraphTraversalParameterInit(&traversal_param);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalParameterInit: %d\n", status);
        exit(0);
     }
 
     status = nvgraphTraversalSetDistancesIndex(&traversal_param, distances_index); 
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalSetDistancesIndex: %d\n", status);
        exit(0);
     }
    
     status = nvgraphTraversalSetPredecessorsIndex(&traversal_param, predecessors_index);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalSetPredecessorsIndex: %d\n", status);
        exit(0);
     }
 
     status = nvgraphTraversalSetUndirectedFlag(&traversal_param, false);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalSetUndirectedFlag: %d\n", status);
        exit(0);
     }
 
     printf("Graph Traverse set \n");
     
     status = nvgraphTraversal(d_graph_handle, d_desc, NVGRAPH_TRAVERSAL_BFS, &starter_vert, traversal_param);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversal: %d\n", status);
        exit(0);
     }
     
     hipEvent_t traverse_time = get_time();
 
     // Get result
     status = nvgraphGetVertexData(d_graph_handle, d_desc, (void*)host_distances, distances_index);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversal: %d\n",status);
        exit(0);
     }

     status = nvgraphGetVertexData(d_graph_handle, d_desc, (void*)host_predecessors, predecessors_index);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversal: %d\n",status);
        exit(0);
     }
 
     hipEvent_t data_retr_time = get_time();
 
     // Print distances for every verticies
     for (int iter1 = 0; iter1 < verticies; iter1++)  
     {
         printf("Distance to vertex %d: %i\n", iter1, host_distances[iter1]);
         printf("Predecessor of vertex %d: %i\n",iter1, host_predecessors[iter1]);
     }
    
 
     free(dimT);
     free(d_input);
     nvgraphDestroyGraphDescr (d_graph_handle, d_desc);
     nvgraphDestroy (d_graph_handle);
 
     
     hipEvent_t end = get_time();
     hipEventSynchronize(end);
 
     free(host_offsets);
     free(host_dest);
     free(host_distances);
 
     float graph_alloc = 0.0;
     hipEventElapsedTime(&graph_alloc, start, create_graph);
 
     float traverse_setup = 0.0;
     hipEventElapsedTime(&traverse_setup, create_graph, traverse_time);
 
     float result_ret_time = 0.0;
     hipEventElapsedTime(&result_ret_time, traverse_time, data_retr_time);
 
     float total_time;
     hipEventElapsedTime(&total_time, start, end);
 
     hipEventDestroy(start);
     hipEventDestroy(create_graph);
     hipEventDestroy(traverse_time);
     hipEventDestroy(data_retr_time);
     hipEventDestroy(end);
 
     printf("Time to create graph in memory  %f \n", graph_alloc);
     printf("Run time traverse setup and BFS %f \n", traverse_setup);
     printf("Time to copy memory back to host %f \n", result_ret_time);
 
     return total_time;
 }