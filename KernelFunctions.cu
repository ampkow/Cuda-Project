#include "hip/hip_runtime.h"
// Contains Kernel functions for Cuda
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>

// STL
#include <vector> 
#include <list> 
#include <iostream>

// Internal Headers
#include "KernelFunctions.h"

// taken from global_memory.cu, Creates event and records time
__host__ hipEvent_t get_time(void)
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
	return time;
}

__global__ void FindShortestPath(int *path,
                                 int *pred,
                                 int  fullSize,
                                 int  dest)
{
    int pointer = dest;

    int pathSize = 1;

    while (pred[pointer] != -1) 
    { 
       path[pathSize - 1] = pred[pointer]; 
       pointer = pred[pointer];
       pathSize++;
    }       
}

// Run BFS algorithm using thurst library
float RunBFSUsingThrust(std::vector<std::vector<int> > &graph,
                         int                            destination,
                         int                            source,
                         int                            totalEdges)
{
    std::list<int> nextVertList; 

    int vertexSize = graph.size();

    hipEvent_t start = get_time();

    thrust::device_vector<int> d_visited(vertexSize);
    thrust::device_vector<int> d_predecessors(vertexSize);
    thrust::device_vector<int> d_distances(vertexSize);
    thrust::device_vector<int> d_path(totalEdges);

    thrust::fill(d_visited.begin(), d_visited.end(), false);
    thrust::fill(d_distances.begin(), d_distances.end(), 0);
    thrust::fill(d_predecessors.begin(), d_predecessors.end(), -1);

    nextVertList.push_back(source);
    d_visited[source] = true;

    hipEvent_t memFinished = get_time();

    bool foundDest = false;

    // BFS algorithm  
    while (!nextVertList.empty()) 
    {  
        int currVertIter = nextVertList.front();  // Current Vertex
        nextVertList.pop_front(); 

        int edgeCount = graph.at(currVertIter).size();

        // need to populate with vector of edges for current vertex
        thrust::device_vector<int> d_edges(graph.at(currVertIter));

        for (int iter0 = 0; iter0 < edgeCount; iter0++)
        {
            int nextVert = d_edges[iter0];
            if (d_visited[nextVert] == false) 
            { 
                printf("Element (%d, %d) = %d \n", currVertIter, iter0, nextVert);
    
                d_visited[nextVert] = true; 
                d_distances[nextVert]    = d_distances[currVertIter] + 1; 
                d_predecessors[nextVert]    = currVertIter; 
    
                nextVertList.push_back(nextVert);
    
                // Stop When finding destination
                if (nextVert == destination) 
                {
                    foundDest = true;
                    break; 
                }
            }
        }

        if (foundDest)
        {
            d_path.push_back(destination);

            int pointer = destination;

            int pathSize = 1;

            // FindShortestPath<<<1, 1>>>(thrust::raw_pointer_cast(d_path.data()),
            //                            thrust::raw_pointer_cast(d_predecessors.data()),
            //                            destination,
            //                            d_path.size());

            while (d_predecessors[pointer] != -1) 
            { 
               pathSize++;
               d_path.push_back(d_predecessors[pointer]); 
               pointer = d_predecessors[pointer];
            }       
              
            // printing path from source to destination 
            printf("\nShortest Path Length is %d\n", pathSize); 
            int count = 0;
            int iter = d_path.size() - 1;
            while (count < pathSize) 
            {
                std::cout << d_path[iter] << " "; 
                count++;
                iter--;
            }

            hipEvent_t bfsFinised = get_time();

            return true;
        }
    } 

    hipEvent_t bfsFailFinised = get_time();

    return false; 
}

__global__ void BFSLevels(int  *vertices,
                          int  *edges,
                          int  *distances,
                          int  *predecessors,
                          int  *vertIndices,
                          int  *edgeLengths,
                          bool *levels,
                          bool *visitedVertices,
                          bool *foundDest,
                          int   numVert,
                          int   destination)
{
    int thrID = threadIdx.x + blockIdx.x * blockDim.x;


    __shared__ bool destFound;
    destFound = false;
    if (thrID < numVert && !destFound)
    {
        int curVert = vertices[thrID];
        //printf("Current Vert %d\n", curVert);
        if (levels[curVert])
        {
            levels[curVert]          = false;
            visitedVertices[curVert] = true;

            printf("CurVert %d\n", curVert);

            int edgesBegin = vertIndices[thrID];
            int edgesEnd   = edgeLengths[thrID] + edgesBegin;  

            for (int edgeIter = edgesBegin; edgeIter < edgesEnd; ++edgeIter)
            {
               int nextVert = edges[edgeIter];
                if (!visitedVertices[nextVert])
                {
                    printf("Vert %d, Edge %d, Begin %d, End %d\n", curVert, edgeIter, edgesBegin, edgesEnd);
             
                    distances[nextVert] = distances[curVert] + 1;
                    levels[nextVert] = true;

                    printf("Vertices %d, edge %d\n", curVert, nextVert);
                    predecessors[nextVert]  = curVert; 
      
                    // Stop When finding destination
                    if (nextVert == destination) 
                    {
                        *foundDest = true;
                        destFound  = true;
                        __syncthreads();
                        printf("Won\n");
                    }
                }
            }
        }
    }    
}

float BFSByLevel(std::vector<int> &vertices,
                 std::vector<int> &edges,
                 std::vector<int> &vertIndices,
                 std::vector<int> &edgeLength,
                 int               destination,
                 int               source)
{
    int numVertices = vertices.size();

    int arraySizeInBytes     = sizeof(int) * numVertices;
    int arraySizeInBytesBool = sizeof(bool) * numVertices;

    // Create Host Arrays to pass into kernel call for BFS
    bool *h_visitedVertices;
    bool *h_levels;
    int  *h_distances;
    int  *h_predecessors;

    h_distances        = (int *)  malloc(arraySizeInBytes); 
    h_predecessors     = (int *)  malloc(arraySizeInBytes);
    h_visitedVertices  = (bool *) malloc(arraySizeInBytesBool); 
    h_levels           = (bool *) malloc(arraySizeInBytesBool); 

    for (int vertexIter = 0; vertexIter < numVertices; ++vertexIter)
    {
        h_visitedVertices[vertexIter] = false;
        h_levels[vertexIter]          = false;
        h_predecessors[vertexIter]    = -1;
    }

    h_levels[source]    = true;
    h_distances[source] = 0;

    bool *d_visitedVertices;
    bool *d_levels;
    int  *d_distances;
    int  *d_predecessors;

    thrust::device_vector<int> d_vertices(vertices);
    thrust::device_vector<int> d_edges(edges);
    thrust::device_vector<int> d_vertIndices(vertIndices);
    thrust::device_vector<int> d_edgeLength(edgeLength);

    hipMalloc((void**) &d_distances,       arraySizeInBytes);
    hipMalloc((void**) &d_predecessors,       arraySizeInBytes);
    hipMalloc((void**) &d_levels,          arraySizeInBytesBool);
    hipMalloc((void**) &d_visitedVertices, arraySizeInBytesBool);

    hipMemcpy(d_levels,           h_levels,           arraySizeInBytesBool, hipMemcpyHostToDevice);
    hipMemcpy(d_visitedVertices,  h_visitedVertices,  arraySizeInBytesBool, hipMemcpyHostToDevice);
    hipMemcpy(d_predecessors,     h_predecessors,     arraySizeInBytes,     hipMemcpyHostToDevice);
    hipMemcpy(d_distances,        h_distances,        arraySizeInBytes,     hipMemcpyHostToDevice);


    int blockCount = 1;
    int numThreads = numVertices;

    bool *h_foundDest;

    hipHostAlloc((void**)&h_foundDest, sizeof(bool), hipHostMallocDefault);

    h_foundDest = false;

    bool *d_foundDest;

    hipMalloc((void**) &d_foundDest,    sizeof(bool));
    hipMemcpy(d_foundDest,  &h_foundDest,  sizeof(bool), hipMemcpyHostToDevice);

    // how to get predecessors???
    int runCount = 0;
    while (runCount < numVertices)
    {
        BFSLevels<<<blockCount, numThreads>>>(thrust::raw_pointer_cast(d_vertices.data()),
                                              thrust::raw_pointer_cast(d_edges.data()),
                                              d_distances,
                                              d_predecessors,
                                              thrust::raw_pointer_cast(d_vertIndices.data()),
                                              thrust::raw_pointer_cast(d_edgeLength.data()),
                                              d_levels,
                                              d_visitedVertices,
                                              d_foundDest,
                                              numVertices,
                                              destination);

        runCount++;
    }

    hipMemcpy(&h_foundDest, d_foundDest, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(h_distances, d_distances, arraySizeInBytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_predecessors, d_predecessors, arraySizeInBytes, hipMemcpyDeviceToHost);

    hipFree(d_distances);
    hipFree(d_levels);
    hipFree(d_visitedVertices);

    for (int distIter = 0; distIter < numVertices; ++distIter)
    {
        printf ("%d   ", h_distances[distIter]);
    }
    printf("\n");

    for (int predIter = 0; predIter < numVertices; ++predIter)
    {
        printf ("%d   ", h_predecessors[predIter]);
    }
    printf("\n");

    thrust::device_vector<int> d_path(edges);

    if (true)
    {
        d_path.push_back(destination);
        int pointer = destination;

            int pathSize = 1;

            while (h_predecessors[pointer] != -1) 
            { 
               pathSize++;
               d_path.push_back(h_predecessors[pointer]); 
               pointer = h_predecessors[pointer];
            }       
              
            // printing path from source to destination 
            printf("\nShortest Path Length is %d\n", pathSize); 
            int count = 0;
            int iter = d_path.size() - 1;
            while (count < pathSize) 
            {
                std::cout << d_path[iter] << " "; 
                count++;
                iter--;
            }
        }

    return 0.0;

}