#include "hip/hip_runtime.h"
// Make Kernel Functions inside of here
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


#include "nvgraph.h"

// STL
#include <vector> 
#include <iostream>

// Internal Headers
#include "KernelFunctions.h"

// Run BFS algorithm using thurst library

// __host__ allows to run on device and host and void code duplication
// taken from global_memory.cu
__host__ hipEvent_t get_time(void)
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
	return time;
}

// utility function to form edge between two vertices 
// source and dest 
// void add_edge(std::vector<std::vector<int> > &graph, int src, int dest) 
// { 
//     graph[src].push_back(dest); 

//     graph[dest].push_back(src); 
// } 
  

__global__ void BFSThrust(int  *graph, 
                          int  *numEdges,     // arrray for number of edges for each vertex
                          int  *queue,        // queue of edges to be searched
                          bool *visited,      // array to see if edges have been visited
                          int   source,
                          int   dest,
                          int   vertexSize,
                          int   maxEdges,
                          int  *pred,
                          int  *dist,
                          bool  &found,
                          int   totalEdges)    // Total number of edges in the graph
{

   //printf("We inside kernel %d, %d, %d\n", graph[0 + 0 + maxEdges], graph[2 + 2 + maxEdges], graph[1 + 1 + maxEdges]);
   for (int iter = 0; iter < vertexSize; iter++) 
   { 
       visited[iter] = false; 
       dist[iter]    = 0; 
       pred[iter]    = -1; 

       //printf("Edges Count %d\n", numEdges[iter]);
   } 
 
  // visited[source] = true; 
   dist[source] = 0; 
   queue[0] = source; 
 
   // BFS algorithm  

   // Pointer for front of Queue
   int pointer     = 0;
   int backPointer = 1;


//    int x = blockIdx.x * blockDim.x + threadIdx.x; //using 2-D location in matrix
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
   //int length = gridDim.x*blockDim.x; //width of a row
   while (pointer < totalEdges) 
   {  
       int queueIter = queue[pointer]; 
       pointer++; 
      //printf("Edges %d\n", numEdges[queueIter]);
       for (int iter = 0; iter < numEdges[queueIter]; iter++) 
       { 
           int index = iter * maxEdges + queueIter;
           printf("Element (%d, %d) = %d\n", queueIter, iter, graph[index]);
           if (visited[graph[index]] == false && graph[index] != -100) 
           {    
              // printf("We inside visit\n");

               visited[graph[index]] = true; 
               dist[graph[index]]    = dist[queueIter] + 1; 
               pred[graph[index]]    = queueIter; 

               queue[backPointer] = graph[index]; 
               backPointer++;
 
               // Stop When finding destination
               if (graph[index] == dest) 
               {
                  found =  true; 
                  printf("WE WON!!!!\n");
                  return;
               }
           } 
       } 
   } 
}

__global__ void FindShortestPath(int *path,
                                 int *pred,
                                 int  dest)
{
    int pointer = dest;

    int pathSize = 0;
    for ( ;pred[pointer] != -1; pathSize++) 
    { 
        path[pathSize] = pred[pointer]; 
        pointer = pred[pointer]; 
    } 

    // printing path from source to destination 
    // std::cout << "\nShortest Path: \n"; 
    // for (int iter = pathSize - 1; iter >= 0; iter--) 
    // {
    //     std::cout << path[iter] << " "; 
    // }

}

int **ConvertVectorTo2D(std::vector<std::vector<int> >  &graph,
                        int                              maxEdges)
{
    int **h_graph;
    int graphSize = graph.size();
    h_graph = new int*[graph.size()];

    h_graph[0] = new int[graphSize * maxEdges];

    for(int iter1 = 1; (iter1 < graph.size()); ++iter1)
    { 
       h_graph[iter1] = h_graph[iter1 - 1] + maxEdges;
    }

    for (int outIter = 0; outIter < graphSize; outIter++)
    {
       for(int iter2 = 0; (iter2 < maxEdges); ++iter2)
       {
        if ( iter2 >= graph[outIter].size())
        {
            h_graph[outIter][iter2] = -100;
        }
        else
        {
            h_graph[outIter][iter2] = graph[outIter][iter2];
        }
       } 
    }

    return h_graph;
}

float RunBFSShortestDistance(std::vector<std::vector<int> > &graph,
                             int                             dest,
                             int                             source,
                             int                             totalEdges)
{
    bool foundDest = false;

    int numVerticies = graph.size();

    int arraySizeVertices = sizeof(int) * numVerticies;
    int arraySizeEdges    = sizeof(int) * totalEdges;

    int maxEdges = 0;

    int *h_numEdges = (int *) malloc(arraySizeVertices); 

    for (int vectIter = 0; vectIter < numVerticies; vectIter++)
    {
       int edgeCount = graph[vectIter].size();
       h_numEdges[vectIter] = edgeCount;

       if (edgeCount > maxEdges)
       {
          maxEdges = edgeCount;
       }
    }

    printf("Converting Vectors of Vectors...\n");

    // Convert Vector of Vector to 2d array
    int** h_graph = ConvertVectorTo2D(graph, maxEdges);


    int graphSize = sizeof(int) * graph.size() + maxEdges;

    printf("Done... %d\n", graphSize);

    bool *d_vertexVisited;   // array to see if edges have been visited
    int  *d_numEdges;        // arrray for number of edges for each vertex
    int  *d_edgeQueue;       // queue of edges to be searched
    int  *d_pred;            // array to store predecssors
    int  *d_dist;            // array to store distances

    int  *d_graph;

    printf("Before allocating graph\n");

    hipMalloc((void**) &d_graph, graphSize);

    printf("Afer allocating graph\n");

    // Allocate  Global memory
    hipMalloc((void**) &d_vertexVisited,  sizeof(bool) * numVerticies); //doesnt need populating
    hipMalloc((void**) &d_numEdges,       arraySizeVertices);            // does
    hipMalloc((void**) &d_edgeQueue,      arraySizeEdges);            // doesnt
    hipMalloc((void**) &d_pred,           arraySizeVertices);            // doesnt
    hipMalloc((void**) &d_dist,           arraySizeVertices);            // doesnt

    // After Cuda Malloc

    printf("After cuda malloc \n");



    hipMemcpy(d_numEdges, h_numEdges, arraySizeVertices, hipMemcpyHostToDevice);

    
    printf("After Edge Malloc \n");

    // allocate correctly and pass to kernel by making verticies pointers so memory on memory
    hipMemcpy(d_graph, h_graph[0], graphSize, hipMemcpyHostToDevice);

    printf("After Copy Graph \n");

    //thrust::device_vector<int> d_graph(numVerticies * totalEdges);
    //thrust::device_vector<int> d_graph(vertices);

    // thrust::copy(&(vertices[0][0]), &(vertices[numVerticies - 1][numVerticies - 1]), d_graph.begin());
    // printf("Afer copy\n");
    // thrust::sequence(d_graph.begin(), d_graph.end());

    printf("Before Kernel Call %d, %d, %d\n", h_graph[0][0], h_graph[0][1], h_graph[1][1] );

    // BFS call
    BFSThrust<<<1,1>>>(d_graph /*thrust::raw_pointer_cast(d_graph.data())*/,
                       d_numEdges,
                       d_edgeQueue, 
                       d_vertexVisited, 
                       source,
                       dest,
                       numVerticies,
                       maxEdges,
                       d_pred,
                       d_dist,
                       foundDest,
                       totalEdges);


    printf("After Kernel Call\n");
    hipDeviceSynchronize();

    hipFree(d_vertexVisited);
    hipFree(d_numEdges);
    hipFree(d_edgeQueue);
    hipFree(d_pred);
    hipFree(d_dist);

    free(h_numEdges);
    //free(vertices);

    return 0.0;
}

 // Runs BFS on a generated graph and prints out
 float run_nvgraph_search(int numVerticies)
 {
     // Graph is in CSR format
     hipEvent_t start = get_time();
 
     int verticies = numVerticies;
     int num_edges = numVerticies;
 
     printf("Num_edges %d\n", num_edges);
     int vertex_numsets = 2; 
 
     // Create Verticies
     int offsets_size_in_bytes = sizeof(int) * (verticies + 1);
     int *host_offsets = (int *) malloc(offsets_size_in_bytes); 
 
     int dest_size_in_bytes = sizeof(int) * (num_edges);
     int *host_dest = (int *) malloc(dest_size_in_bytes); 
 
     for (int offsetIter = 0; offsetIter < verticies; offsetIter++)
     {
         host_offsets[offsetIter] = offsetIter;
     }
 
     // Set Up Destinations
     for (int destIter = 0; destIter < num_edges; destIter++)
     {
         host_dest[destIter] = host_offsets[destIter + 1];
     }

     // source_offsets Array of size nvertices+1, where i element equals to the number of the first edge for this vertex in the list of all outgoing edges in the destination_indices array. Last element stores total number of edges
     //     int source_offsets_h[] = {0, 1, 3, 4, 6, 8, 10, 12};
     //     int destination_indices_h[] = {5, 0, 2, 0, 4, 5, 2, 3, 3, 4, 1, 5};
     // destination_indices 	Array of size nedges, where each value designates destanation vertex for an edge. 
     
     // Last Value of Offsets equal number of edges in graph
     host_offsets[verticies] = num_edges;
 
     // holds results
     int *host_distances    = (int *) malloc(offsets_size_in_bytes);
     int *host_predecessors = (int *) malloc(offsets_size_in_bytes);
 
     // nvgraph values
     nvgraphHandle_t d_graph_handle;
     nvgraphGraphDescr_t d_desc;
     nvgraphCSRTopology32I_t d_input;
     hipDataType* dimT;
     int distances_index = 0;
     int predecessors_index = 1;
     dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
     dimT[distances_index] = HIP_R_32I;
     dimT[predecessors_index] = HIP_R_32I;
 
     //Create Graph
     nvgraphStatus_t status = nvgraphCreate (&d_graph_handle);
     if ((int)status != 0)   
     {
         printf("ERROR nvgraphCreate: %d\n", status);
         exit(0);
     }
 
     status = nvgraphCreateGraphDescr (d_graph_handle, &d_desc);
     if ((int)status != 0)   
     {
         printf("ERROR nvgraphCreateGraphDescr: %d\n", status);
         exit(0);
     }
 
     // Set graph properties
     d_input = (nvgraphCSRTopology32I_t) malloc(sizeof(struct nvgraphCSRTopology32I_st));
     d_input->nvertices = verticies;
     d_input->nedges = num_edges;
     d_input->source_offsets = host_offsets;
     d_input->destination_indices = host_dest;
 
     status = nvgraphSetGraphStructure(d_graph_handle, d_desc, (void*)d_input, NVGRAPH_CSR_32);
     if ((int)status != 0)   
     {
         printf("ERROR nvgraphSetGraphStructure: %d\n", status);
         exit(0);
     }
     
     status = nvgraphAllocateVertexData(d_graph_handle, d_desc, vertex_numsets, dimT);
 
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphAllocateVertexData: %d\n", status);
        exit(0);
     }
 
     hipEvent_t create_graph = get_time();
 
     int starter_vert = 1;
     // Setting the traverse param
     nvgraphTraversalParameter_t traversal_param;
     status = nvgraphTraversalParameterInit(&traversal_param);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalParameterInit: %d\n", status);
        exit(0);
     }
 
     status = nvgraphTraversalSetDistancesIndex(&traversal_param, distances_index); 
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalSetDistancesIndex: %d\n", status);
        exit(0);
     }
    
     status = nvgraphTraversalSetPredecessorsIndex(&traversal_param, predecessors_index);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalSetPredecessorsIndex: %d\n", status);
        exit(0);
     }
 
     status = nvgraphTraversalSetUndirectedFlag(&traversal_param, false);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversalSetUndirectedFlag: %d\n", status);
        exit(0);
     }
 
     printf("Graph Traverse set \n");
     
     status = nvgraphTraversal(d_graph_handle, d_desc, NVGRAPH_TRAVERSAL_BFS, &starter_vert, traversal_param);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversal: %d\n", status);
        exit(0);
     }
     
     hipEvent_t traverse_time = get_time();
 
     // Get result
     status = nvgraphGetVertexData(d_graph_handle, d_desc, (void*)host_distances, distances_index);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversal: %d\n",status);
        exit(0);
     }

     status = nvgraphGetVertexData(d_graph_handle, d_desc, (void*)host_predecessors, predecessors_index);
     if ((int)status != 0)   
     {
        printf("ERROR nvgraphTraversal: %d\n",status);
        exit(0);
     }
 
     hipEvent_t data_retr_time = get_time();
 
     // Print distances for every verticies
     for (int iter1 = 0; iter1 < verticies; iter1++)  
     {
         printf("Distance to vertex %d: %i\n", iter1, host_distances[iter1]);
         printf("Predecessor of vertex %d: %i\n",iter1, host_predecessors[iter1]);
     }
    
 
     free(dimT);
     free(d_input);
     nvgraphDestroyGraphDescr (d_graph_handle, d_desc);
     nvgraphDestroy (d_graph_handle);
 
     
     hipEvent_t end = get_time();
     hipEventSynchronize(end);
 
     free(host_offsets);
     free(host_dest);
     free(host_distances);
 
     float graph_alloc = 0.0;
     hipEventElapsedTime(&graph_alloc, start, create_graph);
 
     float traverse_setup = 0.0;
     hipEventElapsedTime(&traverse_setup, create_graph, traverse_time);
 
     float result_ret_time = 0.0;
     hipEventElapsedTime(&result_ret_time, traverse_time, data_retr_time);
 
     float total_time;
     hipEventElapsedTime(&total_time, start, end);
 
     hipEventDestroy(start);
     hipEventDestroy(create_graph);
     hipEventDestroy(traverse_time);
     hipEventDestroy(data_retr_time);
     hipEventDestroy(end);
 
     printf("Time to create graph in memory  %f \n", graph_alloc);
     printf("Run time traverse setup and BFS %f \n", traverse_setup);
     printf("Time to copy memory back to host %f \n", result_ret_time);
 
     return total_time;
 }